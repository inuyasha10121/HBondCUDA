#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
//#define BENCHMARK_TIMING

#include <stdio.h>
#include <iostream>
#include <math.h>
#include <float.h>
#include <algorithm>

#include "kernel.cuh"

using namespace std;

__global__ void waterFilterKernel(char *out, const GPUAtom *inWaters, const float centx, const float centy, const float centz, const float maxdist, const size_t nWaters)
{
    //Find where we are in the GPU
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    //Make sure we aren't trying to access outside our pre-definied dimensions
    if (i < nWaters)
    {
        //Get the distance between the water and the center of geometry
        float distx = inWaters[i].x - centx;
        float disty = inWaters[i].y - centy;
        float distz = inWaters[i].z - centz;
        float dist = sqrtf((distx * distx) + (disty * disty) + (distz * distz));
        //Flag if it is within the proper distance or not
        out[i] = (dist < maxdist);
    }
}

__global__ void bondDistKernel(char *out, const GPUAtom *inProtein, const GPUAtom *inWaters, const size_t nProteins, const size_t nWaters)
{
    //Find where we are in the GPU
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    //Make sure we aren't trying to access outside our pre-definied dimensions
    if (i < nWaters && j < nProteins)
    {
        //out[(j * nWaters) + i] = 'n';  //Set default to "No bond"
        if (i % 3 == 0) //Only look for oxygen atoms, which should be every third atom starting at atom index 0
        {
            //Get the distance between the heavy atoms
            float distx = inWaters[i].x - inProtein[j].x;
            float disty = inWaters[i].y - inProtein[j].y;
            float distz = inWaters[i].z - inProtein[j].z;
            float dist = sqrtf((distx * distx) + (disty * disty) + (distz * distz));
            out[(j * nWaters) + i] = (dist < 3.5f);
        }
    }
}

__global__ void waterToAcceptorKernel(char *out, const GPUAtom *inAcceptor, const GPUAtom *inWater, const size_t nAcceptors, const size_t nWaters)
{
    //Find where we are in the GPU
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    //Make sure we aren't trying to access outside our pre-definied dimensions
    if (i < nWaters && j < nAcceptors)
    {
        if (out[(j * nWaters) + i] == true) //Make sure we are in bonding distance from before
        {
            //Find which hydrogen is between the acceptor and the oxygen
            float distx = inWater[i + 1].x - inAcceptor[j].x;
            float disty = inWater[i + 1].y - inAcceptor[j].y;
            float distz = inWater[i + 1].z - inAcceptor[j].z;
            float dist1 = sqrtf((distx * distx) + (disty * disty) + (distz * distz));
            distx = inWater[i + 2].x - inAcceptor[j].x;
            disty = inWater[i + 2].y - inAcceptor[j].y;
            distz = inWater[i + 2].z - inAcceptor[j].z;
            float dist2 = sqrtf((distx * distx) + (disty * disty) + (distz * distz));
            int closestindex = -1;
            if (dist1 < dist2)
            {
                closestindex = i + 1;
            }
            else
            {
                closestindex = i + 2;
            }
            //Calculate the angle parameters
            distx = inWater[i].x - inAcceptor[j].x;
            disty = inWater[i].y - inAcceptor[j].y;
            distz = inWater[i].z - inAcceptor[j].z;
            float a = sqrtf((distx * distx) + (disty * disty) + (distz * distz));
            distx = inWater[i].x - inWater[closestindex].x;
            disty = inWater[i].y - inWater[closestindex].y;
            distz = inWater[i].z - inWater[closestindex].z;
            float b = sqrtf((distx * distx) + (disty * disty) + (distz * distz));
            distx = inWater[closestindex].x - inAcceptor[j].x;
            disty = inWater[closestindex].y - inAcceptor[j].y;
            distz = inWater[closestindex].z - inAcceptor[j].z;
            float c = sqrtf((distx * distx) + (disty * disty) + (distz * distz));
            float theta = (acosf(((a * a) + (b*b) - (c*c)) / (2 * a * b))) * (180.0f / M_PI);
            if (theta > 30.0f) //If the angle is too large, change the bond to not a bond
            {
                out[(j * nWaters) + i] = false;
            }
        }
    }
}


__global__ void donorToWaterKernel(char *out, const GPUAtom *inDonor, const GPUAtom *inLinker, const GPUAtom *inWater, const size_t nDonors, const size_t nLinkers, const size_t nWaters)
{
    //Find where we are in the GPU
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    //Make sure we aren't trying to access outside our pre-definied dimensions
    if (i < nWaters && j < nDonors)
    {
        if (out[(j * nWaters) + i] == true) //Make sure we are in bonding distance from before
        {
            int closestindex = -1;
            float mindist = FLT_MAX;  //Equivalent to "c"
            //Find the bridging linker hydrogen in the residue
            for (int k = 0; k < nLinkers; k++)
            {
                if (inLinker[k].resid == inDonor[j].resid) //Hydrogen belongs to same residue
                {
                    float distx = inLinker[k].x - inWater[i].x;
                    float disty = inLinker[k].y - inWater[i].y;
                    float distz = inLinker[k].z - inWater[i].z;
                    float dist = sqrtf((distx * distx) + (disty * disty) + (distz * distz));
                    if (dist < mindist)
                    {
                        mindist = dist;
                        closestindex = k;
                    }
                    else if (inDonor[j].resid < inLinker[k].resid)
                    {
                        break;
                    }
                }
            }
            //Calculate the angle parameter
            float distx = inLinker[closestindex].x - inDonor[j].x;
            float disty = inLinker[closestindex].y - inDonor[j].y;
            float distz = inLinker[closestindex].z - inDonor[j].z;
            float a = sqrtf((distx * distx) + (disty * disty) + (distz * distz));
            distx = inWater[i].x - inDonor[j].x;
            disty = inWater[i].y - inDonor[j].y;
            distz = inWater[i].z - inDonor[j].z;
            float b = sqrtf((distx * distx) + (disty * disty) + (distz * distz));
            float theta = (acosf(((a * a) + (b*b) - (mindist*mindist)) / (2 * a * b))) * (180.0f / M_PI);
            if (theta > 30.0f) //If the angle is too large, change the bond to not a bond
            {
                out[(j * nWaters) + i] = false;
            }
        }
    }
}

__global__ void timelineMapKernel2D(char * outMap, int * timeline, int * tllookup, int * boundAAs, int * boundwaters, const int window, const int threshold, const int nframes, const int nAAs, const int nwaters)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; //Frame
    int j = blockIdx.y * blockDim.y + threadIdx.y; //AA

    if (i < (nframes * nwaters) && j < nAAs)
    {
        int currwater = i / nframes;
        int currframe = i % nframes;

        int boundframes = 0;
        if (currframe < (nframes - window))
        {
            for (int currwindow = 0; currwindow < window; currwindow++)
            {
                
                for (int currsearch = tllookup[currframe + currwindow]; currsearch < tllookup[currframe + currwindow + 1]; currsearch += 2)
                {
                    if ((timeline[currsearch] == boundAAs[j]) && (timeline[currsearch + 1] == boundwaters[currwater]))
                    {
                        boundframes++;
                    }
                }
            }
        }
        outMap[(j * nframes) + i] = (boundframes >= threshold);
    }
}

__global__ void timelineMapKernel1D(char * outMap, int * timeline, int * tllookup, int * boundAAs, int * boundwaters, const int window, const int threshold, const int nframes, const int nAAs, const int nwaters)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < ((nframes - window) * nwaters * nAAs))
    {
        int frame = i / (nwaters * nAAs);
        int water = (i % (nwaters * nAAs)) / nwaters;
        int AA = (i % (nwaters * nAAs)) % nwaters;
        int boundframes = 0;
        for (int currwindow = 0; currwindow < window; currwindow++)
        {
            for (int currsearch = tllookup[frame + currwindow]; currsearch < tllookup[frame + currwindow + 1]; currsearch += 2)
            {
                if ((timeline[currsearch] == boundAAs[AA]) && (timeline[currsearch + 1] == boundwaters[water]))
                {
                    boundframes++;
                }
            }
        }
        outMap[i] = (boundframes >= threshold);
    }
}


__global__ void visitAndBridgerAnalysisKernel(char * outbridger, char * outvisitlist, int * outframesbound, const char * timelinemap, const int nframes, const int nAAs)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; //Frame
    if (i < nframes)
    {
        int boundcount = 0;
        for (int j = 0; j < nAAs; j++)
        {
            if (timelinemap[(j*nframes) + i])
            {
                boundcount++;
                outvisitlist[j] = true;  //Might be dangerous
            }
        }

        outbridger[i] = (boundcount > 1);
    }
}

hipError_t waterFilterCuda(char *out, const GPUAtom *inWater, const float centx, const float centy, const float centz, const float maxdist, const size_t nWaters, hipDeviceProp_t &deviceProp)
{
    // the device arrays
    GPUAtom *dev_inWater = 0;
    char *dev_out = 0;
    hipError_t cudaStatus;

    // use div because it's more accurrate than the rounding BS
    auto gridDiv = div(nWaters, deviceProp.maxThreadsPerBlock);
    auto gridY = gridDiv.quot;

    // ass backwards way of rounding up (maybe use the same trick as above? It might be "faster")
    if (gridDiv.rem != 0)
        gridY++;

    // find the block and grid size
    auto blockSize = deviceProp.maxThreadsPerBlock;
    int gridSize = min(16 * deviceProp.multiProcessorCount, gridY);

    // Allocate GPU buffers for vectors
    cudaStatus = hipMalloc((void**)&dev_out, nWaters * sizeof(char));
    if (cudaStatus != hipSuccess) {
        cerr << "hipMalloc failed!" << endl;
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_inWater, nWaters * sizeof(GPUAtom));
    if (cudaStatus != hipSuccess) {
        cerr << "hipMalloc failed!" << endl;
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_inWater, inWater, nWaters * sizeof(GPUAtom), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed!" << endl;
        goto Error;
    }

    // Launch a kernel on the GPU.
    waterFilterKernel << <gridSize, blockSize >> > (dev_out, dev_inWater, centx, centy, centz, maxdist, nWaters);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        cerr << "dielectric kernel launch failed: " << hipGetErrorString(cudaStatus) << endl;
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        cerr << "hipDeviceSynchronize returned error code " << cudaStatus << " after launching density kernel!" << endl;
        cout << "Cuda failure " << __FILE__ << ":" << __LINE__ << " '" << hipGetErrorString(cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(out, dev_out, nWaters * sizeof(char), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed!" << endl;
        goto Error;
    }

    // delete all our device arrays
Error:
    hipFree(dev_inWater);
    hipFree(dev_out);

    return cudaStatus;
}

hipError_t bondDistCuda(char *out, const GPUAtom *inProteins, const GPUAtom *inWaters, const size_t nProteins, const size_t nWaters, hipDeviceProp_t &deviceProp)
{
    // define device arrays
    GPUAtom *dev_protein = 0;
    GPUAtom *dev_water = 0;
    char *dev_out = 0;
    hipError_t cudaStatus;

    // Setup the kernel dimensions
    int blockDim = sqrt(deviceProp.maxThreadsPerBlock);
    auto blockSize = dim3(blockDim, blockDim);
    //Waters are chosen for x dimension, since CUDA can handle MUCH more data along the x dimension than y.
    auto gridSize = dim3(round((blockDim - 1 + nWaters) / blockDim), round((blockDim - 1 + nProteins) / blockDim));

    // Allocate GPU buffers for vectors.
    cudaStatus = hipMalloc((void**)&dev_out, nProteins * nWaters * sizeof(char));
    if (cudaStatus != hipSuccess) {
        cerr << "hipMalloc failed!" << endl;
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_protein, nProteins * sizeof(GPUAtom));
    if (cudaStatus != hipSuccess) {
        cerr << "hipMalloc failed!" << endl;
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_water, nWaters * sizeof(GPUAtom));
    if (cudaStatus != hipSuccess) {
        cerr << "hipMalloc failed!" << endl;
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_protein, inProteins, nProteins * sizeof(GPUAtom), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed!" << endl;
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_water, inWaters, nWaters * sizeof(GPUAtom), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed!" << endl;
        goto Error;
    }

    // Launch a kernel on the GPU.
    bondDistKernel << <gridSize, blockSize >> > (dev_out, dev_protein, dev_water, nProteins, nWaters);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        cerr << "Distance kernel launch failed: " << hipGetErrorString(cudaStatus) << endl;
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        cerr << "hipDeviceSynchronize returned error code " << cudaStatus << " after launching hbond distance kernel!" << endl;
        cout << "Cuda failure " << __FILE__ << ":" << __LINE__ << " '" << hipGetErrorString(cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(out, dev_out, nProteins * nWaters * sizeof(char), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed!" << endl;
        goto Error;
    }

    // clear all our device arrays
Error:
    hipFree(dev_protein);
    hipFree(dev_water);
    hipFree(dev_out);

    return cudaStatus;
}

hipError_t waterToAcceptorCuda(char *out, const GPUAtom *inAcceptor, const GPUAtom *inWater, const size_t nAcceptors, const size_t nWaters, hipDeviceProp_t &deviceProp)
{
    // define device arrays
    GPUAtom *dev_acceptor = 0;
    GPUAtom *dev_water = 0;
    char *dev_out = 0;
    hipError_t cudaStatus;

    // Setup the kernel dimensions
    int blockDim = sqrt(deviceProp.maxThreadsPerBlock);
    auto blockSize = dim3(blockDim, blockDim);
    //Waters are chosen for x dimension, since CUDA can handle MUCH more data along the x dimension than y.
    auto gridSize = dim3(round((blockDim - 1 + nWaters) / blockDim), round((blockDim - 1 + nAcceptors) / blockDim));

    // Allocate GPU buffers for vectors.
    cudaStatus = hipMalloc((void**)&dev_out, nAcceptors * nWaters * sizeof(char));
    if (cudaStatus != hipSuccess) {
        cerr << "hipMalloc failed!" << endl;
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_acceptor, nAcceptors * sizeof(GPUAtom));
    if (cudaStatus != hipSuccess) {
        cerr << "hipMalloc failed!" << endl;
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_water, nWaters * sizeof(GPUAtom));
    if (cudaStatus != hipSuccess) {
        cerr << "hipMalloc failed!" << endl;
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_out, out, nAcceptors * nWaters * sizeof(char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed!" << endl;
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_acceptor, inAcceptor, nAcceptors * sizeof(GPUAtom), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed!" << endl;
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_water, inWater, nWaters * sizeof(GPUAtom), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed!" << endl;
        goto Error;
    }

    // Launch a kernel on the GPU.
    waterToAcceptorKernel << <gridSize, blockSize >> > (dev_out, dev_acceptor, dev_water, nAcceptors, nWaters);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        cerr << "Donor to water angle kernel launch failed: " << hipGetErrorString(cudaStatus) << endl;
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        cerr << "hipDeviceSynchronize returned error code " << cudaStatus << " after launching donor to water angle kernel!" << endl;
        cout << "Cuda failure " << __FILE__ << ":" << __LINE__ << " '" << hipGetErrorString(cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(out, dev_out, nAcceptors * nWaters * sizeof(char), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed!" << endl;
        goto Error;
    }

    // clear all our device arrays
Error:
    hipFree(dev_acceptor);
    hipFree(dev_water);
    hipFree(dev_out);

    return cudaStatus;
}

hipError_t donorToWaterCuda(char *out, const GPUAtom *inDonor, const GPUAtom *inLinker, const GPUAtom *inWater, const size_t nDonors, const size_t nLinkers, const size_t nWaters, hipDeviceProp_t &deviceProp)
{
    // define device arrays
    GPUAtom *dev_donor = 0;
    GPUAtom *dev_linker = 0;
    GPUAtom *dev_water = 0;
    char *dev_out = 0;
    hipError_t cudaStatus;

    // Setup the kernel dimensions
    int blockDim = sqrt(deviceProp.maxThreadsPerBlock);
    auto blockSize = dim3(blockDim, blockDim);
    //Waters are chosen for x dimension, since CUDA can handle MUCH more data along the x dimension than y.
    auto gridSize = dim3(round((blockDim - 1 + nWaters) / blockDim), round((blockDim - 1 + nDonors) / blockDim));

    // Allocate GPU buffers for vectors.
    cudaStatus = hipMalloc((void**)&dev_out, nDonors * nWaters * sizeof(char));
    if (cudaStatus != hipSuccess) {
        cerr << "hipMalloc failed!" << endl;
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_donor, nDonors * sizeof(GPUAtom));
    if (cudaStatus != hipSuccess) {
        cerr << "hipMalloc failed!" << endl;
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_linker, nLinkers * sizeof(GPUAtom));
    if (cudaStatus != hipSuccess) {
        cerr << "hipMalloc failed!" << endl;
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_water, nWaters * sizeof(GPUAtom));
    if (cudaStatus != hipSuccess) {
        cerr << "hipMalloc failed!" << endl;
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_out, out, nDonors * nWaters * sizeof(char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed!" << endl;
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_donor, inDonor, nDonors * sizeof(GPUAtom), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed!" << endl;
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_linker, inLinker, nLinkers * sizeof(GPUAtom), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed!" << endl;
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_water, inWater, nWaters * sizeof(GPUAtom), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed!" << endl;
        goto Error;
    }

    // Launch a kernel on the GPU.
    donorToWaterKernel << <gridSize, blockSize >> > (dev_out, dev_donor, dev_linker, dev_water, nDonors, nLinkers, nWaters);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        cerr << "Water to acceptor angle kernel launch failed: " << hipGetErrorString(cudaStatus) << endl;
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        cerr << "hipDeviceSynchronize returned error code " << cudaStatus << " after launching water to acceptor angle kernel!" << endl;
        cout << "Cuda failure " << __FILE__ << ":" << __LINE__ << " '" << hipGetErrorString(cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(out, dev_out, nDonors * nWaters * sizeof(char), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed!" << endl;
        goto Error;
    }

    // clear all our device arrays
Error:
    hipFree(dev_donor);
    hipFree(dev_linker);
    hipFree(dev_water);
    hipFree(dev_out);

    return cudaStatus;
}

hipError_t timelineMapCuda2D(char * outMap, const int * timeline, const int * tllookup, const int * boundAAs, const int * boundwaters, const int window, const int threshold, 
    const int ntimeline, const int nframes, const int nAAs, const int nwaters, hipDeviceProp_t &deviceProp)
{
    // define device arrays
    char * dev_outMap = 0;
    int * dev_timeline = 0;
    int * dev_tllookup = 0;
    int * dev_boundAAs = 0;
    int * dev_boundwaters = 0;
    hipError_t cudaStatus;

    //For GPU benchmarking
#ifdef BENCHMARK_TIMING
    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    printf("\n\nSTARTING BENCHMARKING:\n");
#endif
    // Setup the kernel dimensions
    
    int blockDim = sqrt(deviceProp.maxThreadsPerBlock);
    auto blockSize = dim3(blockDim, blockDim);
    //Waters are chosen for x dimension, since CUDA can handle MUCH more data along the x dimension than y.
    auto gridSize = dim3(round((blockDim - 1 + (nframes * nwaters)) / blockDim), round((blockDim - 1 + nAAs) / blockDim));

    /*
    int occBlockSize;
    int occMinGridSize;
    int occGridSize;
    hipOccupancyMaxPotentialBlockSize(&occMinGridSize, &occBlockSize, timelineMapKernel, 0, nwaters * nframes * nAAs);
    occGridSize = ((nwaters * nframes * nAAs) + occBlockSize - 1) / occBlockSize;
    */

    // Allocate GPU buffers for vectors.
#ifdef BENCHMARK_TIMING
    hipEventRecord(start, 0);
#endif
    cudaStatus = hipMalloc((void**)&dev_outMap, nframes * nAAs * nwaters * sizeof(char));
    if (cudaStatus != hipSuccess) {
        cerr << "hipMalloc failed!" << endl;
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_timeline, ntimeline * sizeof(int));
    if (cudaStatus != hipSuccess) {
        cerr << "hipMalloc failed!" << endl;
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_tllookup, (nframes+1) * sizeof(int));
    if (cudaStatus != hipSuccess) {
        cerr << "hipMalloc failed!" << endl;
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_boundAAs, nAAs * sizeof(int));
    if (cudaStatus != hipSuccess) {
        cerr << "hipMalloc failed!" << endl;
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_boundwaters, nwaters * sizeof(int));
    if (cudaStatus != hipSuccess) {
        cerr << "hipMalloc failed!" << endl;
        goto Error;
    }
#ifdef BENCHMARK_TIMING
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Malloc elapsed time:  %3.3f ms \n", time);
    hipEventRecord(start, 0);
#endif

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_timeline, timeline, nframes * nAAs * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed!" << endl;
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_tllookup, tllookup, (nframes+1) * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed!" << endl;
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_boundAAs, boundAAs, nAAs * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed!" << endl;
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_boundwaters, boundwaters, nwaters * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed!" << endl;
        goto Error;
    }
#ifdef BENCHMARK_TIMING
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Memcpy elapsed time:  %3.3f ms \n", time);
    hipEventRecord(start, 0);
#endif

    timelineMapKernel2D << <gridSize, blockSize >> > (dev_outMap, dev_timeline, dev_tllookup, dev_boundAAs, dev_boundwaters, window, threshold, nframes, nAAs, nwaters);
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        cerr << "Timeline map kernel launch failed: " << hipGetErrorString(cudaStatus) << endl;
        cerr << "This sometimes occurs when the graphics card is overloaded." << endl;
        cerr << "Try running again with a lower -gpumem percentage." << endl;
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        cerr << "hipDeviceSynchronize returned error code " << cudaStatus << " after launching timeline map kernel!" << endl;
        cout << "Cuda failure " << __FILE__ << ":" << __LINE__ << " '" << hipGetErrorString(cudaStatus);
        goto Error;
    }

#ifdef BENCHMARK_TIMING
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Kernel+Sync elapsed time:  %3.3f ms \n", time);
    hipEventRecord(start, 0);
#endif


    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(outMap, dev_outMap, nframes * nAAs * nwaters * sizeof(char), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed!" << endl;
        goto Error;
    }
#ifdef BENCHMARK_TIMING
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Memcpy elapsed time:  %3.3f ms \n", time);
#endif

    // clear all our device arrays
Error:
    hipFree(dev_outMap);
    hipFree(dev_timeline);
    hipFree(dev_tllookup);
    hipFree(dev_boundAAs);

    return cudaStatus;
}

hipError_t timelineMapCuda1D(char * outMap, const int * timeline, const int * tllookup, const int * boundAAs, const int * boundwaters, const int window, const int threshold,
    const int ntimeline, const int nframes, const int nAAs, const int nwaters, hipDeviceProp_t &deviceProp)
{
    // define device arrays
    char * dev_outMap = 0;
    int * dev_timeline = 0;
    int * dev_tllookup = 0;
    int * dev_boundAAs = 0;
    int * dev_boundwaters = 0;
    hipError_t cudaStatus;

    //For GPU benchmarking
#ifdef BENCHMARK_TIMING
    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    printf("\n\nSTARTING BENCHMARKING:\n");
#endif
    // Setup the kernel dimensions
    int occBlockSize;
    int occMinGridSize;
    int occGridSize;
    hipOccupancyMaxPotentialBlockSize(&occMinGridSize, &occBlockSize, timelineMapKernel1D, 0, nwaters * nframes * nAAs);
    occGridSize = ((nwaters * nframes * nAAs) + occBlockSize - 1) / occBlockSize;

    // Allocate GPU buffers for vectors.
#ifdef BENCHMARK_TIMING
    hipEventRecord(start, 0);
#endif
    cudaStatus = hipMalloc((void**)&dev_outMap, nframes * nAAs * nwaters * sizeof(char));
    if (cudaStatus != hipSuccess) {
        cerr << "hipMalloc failed!" << endl;
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_timeline, ntimeline * sizeof(int));
    if (cudaStatus != hipSuccess) {
        cerr << "hipMalloc failed!" << endl;
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_tllookup, (nframes + 1) * sizeof(int));
    if (cudaStatus != hipSuccess) {
        cerr << "hipMalloc failed!" << endl;
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_boundAAs, nAAs * sizeof(int));
    if (cudaStatus != hipSuccess) {
        cerr << "hipMalloc failed!" << endl;
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_boundwaters, nwaters * sizeof(int));
    if (cudaStatus != hipSuccess) {
        cerr << "hipMalloc failed!" << endl;
        goto Error;
    }
#ifdef BENCHMARK_TIMING
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Malloc elapsed time:  %3.3f ms \n", time);
    hipEventRecord(start, 0);
#endif

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_timeline, timeline, nframes * nAAs * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed!" << endl;
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_tllookup, tllookup, (nframes + 1) * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed!" << endl;
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_boundAAs, boundAAs, nAAs * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed!" << endl;
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_boundwaters, boundwaters, nwaters * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed!" << endl;
        goto Error;
    }
#ifdef BENCHMARK_TIMING
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Memcpy elapsed time:  %3.3f ms \n", time);
    hipEventRecord(start, 0);
#endif

    timelineMapKernel1D << <occGridSize, occBlockSize >> > (dev_outMap, dev_timeline, dev_tllookup, dev_boundAAs, dev_boundwaters, window, threshold, nframes, nAAs, nwaters);
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        cerr << "Timeline map kernel launch failed: " << hipGetErrorString(cudaStatus) << endl;
        cerr << "This sometimes occurs when the graphics card is overloaded." << endl;
        cerr << "Try running again with a lower -gpumem percentage." << endl;
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        cerr << "hipDeviceSynchronize returned error code " << cudaStatus << " after launching timeline map kernel!" << endl;
        cout << "Cuda failure " << __FILE__ << ":" << __LINE__ << " '" << hipGetErrorString(cudaStatus);
        goto Error;
    }

#ifdef BENCHMARK_TIMING
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Kernel+Sync elapsed time:  %3.3f ms \n", time);
    hipEventRecord(start, 0);
#endif


    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(outMap, dev_outMap, nframes * nAAs * nwaters * sizeof(char), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed!" << endl;
        goto Error;
    }
#ifdef BENCHMARK_TIMING
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Memcpy elapsed time:  %3.3f ms \n", time);
#endif

    // clear all our device arrays
Error:
    hipFree(dev_outMap);
    hipFree(dev_timeline);
    hipFree(dev_tllookup);
    hipFree(dev_boundAAs);

    return cudaStatus;
}

hipError_t visitAndBridgerAnalysisCuda(char * outbridger, char * outvisitlist, int * outframesbound, const char * timelinemap, const int nframes, const int nAAs, hipDeviceProp_t &deviceProp)
{
    // the device arrays
    char * dev_outbridger = 0;
    char * dev_outvisitlist = 0;
    int * dev_outframesbound = 0;
    char * dev_timelinemap = 0;

    hipError_t cudaStatus;

    // use div because it's more accurrate than the rounding BS
    auto gridDiv = div(nframes, deviceProp.maxThreadsPerBlock);
    auto gridY = gridDiv.quot;

    // ass backwards way of rounding up (maybe use the same trick as above? It might be "faster")
    if (gridDiv.rem != 0)
        gridY++;

    // find the block and grid size
    auto blockSize = deviceProp.maxThreadsPerBlock;
    int gridSize = min(16 * deviceProp.multiProcessorCount, gridY);

    // Allocate GPU buffers for vectors
    cudaStatus = hipMalloc((void**)&dev_outbridger, nframes * sizeof(char));
    if (cudaStatus != hipSuccess) {
        cerr << "hipMalloc failed!" << endl;
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_outvisitlist, nAAs * sizeof(char));
    if (cudaStatus != hipSuccess) {
        cerr << "hipMalloc failed!" << endl;
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_outframesbound, nframes * sizeof(int));
    if (cudaStatus != hipSuccess) {
        cerr << "hipMalloc failed!" << endl;
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_timelinemap, nframes * nAAs * sizeof(char));
    if (cudaStatus != hipSuccess) {
        cerr << "hipMalloc failed!" << endl;
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_timelinemap, timelinemap, nframes * nAAs * sizeof(char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed!" << endl;
        goto Error;
    }

    // Launch a kernel on the GPU.  (char * outbridger, char * outvisitlist, int * outframesbound, int * outevents, const char * timelinemap, const int nframes, const int nAAs, hipDeviceProp_t &deviceProp)
    visitAndBridgerAnalysisKernel << <gridSize, blockSize >> > (dev_outbridger, dev_outvisitlist, dev_outframesbound, dev_timelinemap, nframes, nAAs);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        cerr << "Visit and bridger analysis kernel launch failed: " << hipGetErrorString(cudaStatus) << endl;
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        cerr << "hipDeviceSynchronize returned error code " << cudaStatus << " after launching visit and bridger analysis kernel!" << endl;
        cout << "Cuda failure " << __FILE__ << ":" << __LINE__ << " '" << hipGetErrorString(cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(outbridger, dev_outbridger, nframes * sizeof(char), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed!" << endl;
        goto Error;
    }

    cudaStatus = hipMemcpy(outvisitlist, dev_outvisitlist, nAAs * sizeof(char), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed!" << endl;
        goto Error;
    }

    cudaStatus = hipMemcpy(outframesbound, dev_outframesbound, nframes * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed!" << endl;
        goto Error;
    }

    // delete all our device arrays
Error:
    hipFree(dev_outbridger);
    hipFree(dev_outvisitlist);
    hipFree(dev_outframesbound);
    hipFree(dev_timelinemap);

    return cudaStatus;
}