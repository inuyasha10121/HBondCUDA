#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES

#include <stdio.h>
#include <iostream>
#include <math.h>
#include <algorithm>

#include "kernel.cuh"

using namespace std;

__global__ void waterFilterKernel(char *out, const GPUAtom *inWaters, const float centx, const float centy, const float centz, const float maxdist, const size_t nWaters)
{
    //Find where we are in the GPU
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    //Make sure we aren't trying to access outside our pre-definied dimensions
    if (i < nWaters)
    {
        //Get the distance between the water and the center of geometry
        float distx = inWaters[i].x - centx;
        float disty = inWaters[i].y - centy;
        float distz = inWaters[i].z - centz;
        float dist = sqrtf((distx * distx) + (disty * disty) + (distz * distz));
        //Flag if it is within the proper distance or not
        out[i] = (dist < maxdist);
    }
}

__global__ void bondDistKernel(char *out, const GPUAtom *inProtein, const GPUAtom *inWaters, const size_t nProteins, const size_t nWaters)
{
    //Find where we are in the GPU
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    //Make sure we aren't trying to access outside our pre-definied dimensions
    if (i < nWaters && j < nProteins)
    {
        //out[(j * nWaters) + i] = 'n';  //Set default to "No bond"
        if (i % 3 == 0) //Only look for oxygen atoms, which should be every third atom starting at atom index 0
        {
            //Get the distance between the heavy atoms
            float distx = inWaters[i].x - inProtein[j].x;
            float disty = inWaters[i].y - inProtein[j].y;
            float distz = inWaters[i].z - inProtein[j].z;
            float dist = sqrtf((distx * distx) + (disty * disty) + (distz * distz));
            out[(j * nWaters) + i] = (dist < 3.5f);
        }
    }
}

__global__ void waterToAcceptorKernel(char *out, const GPUAtom *inAcceptor, const GPUAtom *inWater, const size_t nAcceptors, const size_t nWaters)
{
    //Find where we are in the GPU
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    //Make sure we aren't trying to access outside our pre-definied dimensions
    if (i < nWaters && j < nAcceptors)
    {
        if (out[(j * nWaters) + i] == true) //Make sure we are in bonding distance from before
        {
            //Find which hydrogen is between the acceptor and the oxygen
            float distx = inWater[i + 1].x - inAcceptor[j].x;
            float disty = inWater[i + 1].y - inAcceptor[j].y;
            float distz = inWater[i + 1].z - inAcceptor[j].z;
            float dist1 = sqrtf((distx * distx) + (disty * disty) + (distz * distz));
            distx = inWater[i + 2].x - inAcceptor[j].x;
            disty = inWater[i + 2].y - inAcceptor[j].y;
            distz = inWater[i + 2].z - inAcceptor[j].z;
            float dist2 = sqrtf((distx * distx) + (disty * disty) + (distz * distz));
            int closestindex = -1;
            if (dist1 < dist2)
            {
                closestindex = i + 1;
            }
            else
            {
                closestindex = i + 2;
            }
            //Calculate the angle parameters
            distx = inWater[i].x - inAcceptor[j].x;
            disty = inWater[i].y - inAcceptor[j].y;
            distz = inWater[i].z - inAcceptor[j].z;
            float a = sqrtf((distx * distx) + (disty * disty) + (distz * distz));
            distx = inWater[i].x - inWater[closestindex].x;
            disty = inWater[i].y - inWater[closestindex].y;
            distz = inWater[i].z - inWater[closestindex].z;
            float b = sqrtf((distx * distx) + (disty * disty) + (distz * distz));
            distx = inWater[closestindex].x - inAcceptor[j].x;
            disty = inWater[closestindex].y - inAcceptor[j].y;
            distz = inWater[closestindex].z - inAcceptor[j].z;
            float c = sqrtf((distx * distx) + (disty * disty) + (distz * distz));
            float theta = (acosf(((a * a) + (b*b) - (c*c)) / (2 * a * b))) * (180.0f / M_PI);
            if (theta > 30.0f) //If the angle is too large, change the bond to not a bond
            {
                out[(j * nWaters) + i] = false;
            }
        }
    }
}


__global__ void donorToWaterKernel(char *out, const GPUAtom *inDonor, const GPUAtom *inLinker, const GPUAtom *inWater, const size_t nDonors, const size_t nLinkers, const size_t nWaters)
{
    //Find where we are in the GPU
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    //Make sure we aren't trying to access outside our pre-definied dimensions
    if (i < nWaters && j < nDonors)
    {
        if (out[(j * nWaters) + i] == true) //Make sure we are in bonding distance from before
        {
            int closestindex = -1;
            float mindist = FLT_MAX;  //Equivalent to "c"
            //Find the bridging linker hydrogen in the residue
            for (int k = 0; k < nLinkers; k++)
            {
                if (inLinker[k].resid == inDonor[j].resid) //Hydrogen belongs to same residue
                {
                    float distx = inLinker[k].x - inWater[i].x;
                    float disty = inLinker[k].y - inWater[i].y;
                    float distz = inLinker[k].z - inWater[i].z;
                    float dist = sqrtf((distx * distx) + (disty * disty) + (distz * distz));
                    if (dist < mindist)
                    {
                        mindist = dist;
                        closestindex = k;
                    }
                    else if (inDonor[j].resid < inLinker[k].resid)
                    {
                        break;
                    }
                }
            }
            //Calculate the angle parameter
            float distx = inLinker[closestindex].x - inDonor[j].x;
            float disty = inLinker[closestindex].y - inDonor[j].y;
            float distz = inLinker[closestindex].z - inDonor[j].z;
            float a = sqrtf((distx * distx) + (disty * disty) + (distz * distz));
            distx = inWater[i].x - inDonor[j].x;
            disty = inWater[i].y - inDonor[j].y;
            distz = inWater[i].z - inDonor[j].z;
            float b = sqrtf((distx * distx) + (disty * disty) + (distz * distz));
            float theta = (acosf(((a * a) + (b*b) - (mindist*mindist)) / (2 * a * b))) * (180.0f / M_PI);
            if (theta > 30.0f) //If the angle is too large, change the bond to not a bond
            {
                out[(j * nWaters) + i] = false;
            }
        }
    }
}

hipError_t waterFilterCuda(char *out, const GPUAtom *inWater, const float centx, const float centy, const float centz, const float maxdist, const size_t nWaters, hipDeviceProp_t &deviceProp)
{
    // the device arrays
    GPUAtom *dev_inWater = 0;
    char *dev_out = 0;
    hipError_t cudaStatus;

    // use div because it's more accurrate than the rounding BS
    auto gridDiv = div(nWaters, deviceProp.maxThreadsPerBlock);
    auto gridY = gridDiv.quot;

    // ass backwards way of rounding up (maybe use the same trick as above? It might be "faster")
    if (gridDiv.rem != 0)
        gridY++;

    // find the block and grid size
    auto blockSize = deviceProp.maxThreadsPerBlock;
    int gridSize = min(16 * deviceProp.multiProcessorCount, gridY);

    // Allocate GPU buffers for vectors
    cudaStatus = hipMalloc((void**)&dev_out, nWaters * sizeof(char));
    if (cudaStatus != hipSuccess) {
        cerr << "hipMalloc failed!" << endl;
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_inWater, nWaters * sizeof(GPUAtom));
    if (cudaStatus != hipSuccess) {
        cerr << "hipMalloc failed!" << endl;
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_inWater, inWater, nWaters * sizeof(GPUAtom), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed!" << endl;
        goto Error;
    }

    // Launch a kernel on the GPU.
    waterFilterKernel << <gridSize, blockSize >> > (dev_out, dev_inWater, centx, centy, centz, maxdist, nWaters);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        cerr << "dielectric kernel launch failed: " << hipGetErrorString(cudaStatus) << endl;
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        cerr << "hipDeviceSynchronize returned error code " << cudaStatus << " after launching density kernel!" << endl;
        cout << "Cuda failure " << __FILE__ << ":" << __LINE__ << " '" << hipGetErrorString(cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(out, dev_out, nWaters * sizeof(char), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed!" << endl;
        goto Error;
    }

    // delete all our device arrays
Error:
    hipFree(dev_inWater);
    hipFree(dev_out);

    return cudaStatus;
}

hipError_t bondDistCuda(char *out, const GPUAtom *inProteins, const GPUAtom *inWaters, const size_t nProteins, const size_t nWaters, hipDeviceProp_t &deviceProp)
{
    // define device arrays
    GPUAtom *dev_protein = 0;
    GPUAtom *dev_water = 0;
    char *dev_out = 0;
    hipError_t cudaStatus;

    // Setup the kernel dimensions
    int blockDim = sqrt(deviceProp.maxThreadsPerBlock);
    auto blockSize = dim3(blockDim, blockDim);
    //Waters are chosen for x dimension, since CUDA can handle MUCH more data along the x dimension than y.
    auto gridSize = dim3(round((blockDim - 1 + nWaters) / blockDim), round((blockDim - 1 + nProteins) / blockDim));

    // Allocate GPU buffers for vectors.
    cudaStatus = hipMalloc((void**)&dev_out, nProteins * nWaters * sizeof(char));
    if (cudaStatus != hipSuccess) {
        cerr << "hipMalloc failed!" << endl;
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_protein, nProteins * sizeof(GPUAtom));
    if (cudaStatus != hipSuccess) {
        cerr << "hipMalloc failed!" << endl;
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_water, nWaters * sizeof(GPUAtom));
    if (cudaStatus != hipSuccess) {
        cerr << "hipMalloc failed!" << endl;
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_protein, inProteins, nProteins * sizeof(GPUAtom), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed!" << endl;
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_water, inWaters, nWaters * sizeof(GPUAtom), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed!" << endl;
        goto Error;
    }

    // Launch a kernel on the GPU.
    bondDistKernel << <gridSize, blockSize >> > (dev_out, dev_protein, dev_water, nProteins, nWaters);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        cerr << "Distance kernel launch failed: " << hipGetErrorString(cudaStatus) << endl;
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        cerr << "hipDeviceSynchronize returned error code " << cudaStatus << " after launching hbond distance kernel!" << endl;
        cout << "Cuda failure " << __FILE__ << ":" << __LINE__ << " '" << hipGetErrorString(cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(out, dev_out, nProteins * nWaters * sizeof(char), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed!" << endl;
        goto Error;
    }

    // clear all our device arrays
Error:
    hipFree(dev_protein);
    hipFree(dev_water);
    hipFree(dev_out);

    return cudaStatus;
}

hipError_t waterToAcceptorCuda(char *out, const GPUAtom *inAcceptor, const GPUAtom *inWater, const size_t nAcceptors, const size_t nWaters, hipDeviceProp_t &deviceProp)
{
    // define device arrays
    GPUAtom *dev_acceptor = 0;
    GPUAtom *dev_water = 0;
    char *dev_out = 0;
    hipError_t cudaStatus;

    // Setup the kernel dimensions
    int blockDim = sqrt(deviceProp.maxThreadsPerBlock);
    auto blockSize = dim3(blockDim, blockDim);
    //Waters are chosen for x dimension, since CUDA can handle MUCH more data along the x dimension than y.
    auto gridSize = dim3(round((blockDim - 1 + nWaters) / blockDim), round((blockDim - 1 + nAcceptors) / blockDim));

    // Allocate GPU buffers for vectors.
    cudaStatus = hipMalloc((void**)&dev_out, nAcceptors * nWaters * sizeof(char));
    if (cudaStatus != hipSuccess) {
        cerr << "hipMalloc failed!" << endl;
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_acceptor, nAcceptors * sizeof(GPUAtom));
    if (cudaStatus != hipSuccess) {
        cerr << "hipMalloc failed!" << endl;
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_water, nWaters * sizeof(GPUAtom));
    if (cudaStatus != hipSuccess) {
        cerr << "hipMalloc failed!" << endl;
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_out, out, nAcceptors * nWaters * sizeof(char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed!" << endl;
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_acceptor, inAcceptor, nAcceptors * sizeof(GPUAtom), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed!" << endl;
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_water, inWater, nWaters * sizeof(GPUAtom), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed!" << endl;
        goto Error;
    }

    // Launch a kernel on the GPU.
    waterToAcceptorKernel << <gridSize, blockSize >> > (dev_out, dev_acceptor, dev_water, nAcceptors, nWaters);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        cerr << "Donor to water angle kernel launch failed: " << hipGetErrorString(cudaStatus) << endl;
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        cerr << "hipDeviceSynchronize returned error code " << cudaStatus << " after launching donor to water angle kernel!" << endl;
        cout << "Cuda failure " << __FILE__ << ":" << __LINE__ << " '" << hipGetErrorString(cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(out, dev_out, nAcceptors * nWaters * sizeof(char), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed!" << endl;
        goto Error;
    }

    // clear all our device arrays
Error:
    hipFree(dev_acceptor);
    hipFree(dev_water);
    hipFree(dev_out);

    return cudaStatus;
}

hipError_t donorToWaterCuda(char *out, const GPUAtom *inDonor, const GPUAtom *inLinker, const GPUAtom *inWater, const size_t nDonors, const size_t nLinkers, const size_t nWaters, hipDeviceProp_t &deviceProp)
{
    // define device arrays
    GPUAtom *dev_donor = 0;
    GPUAtom *dev_linker = 0;
    GPUAtom *dev_water = 0;
    char *dev_out = 0;
    hipError_t cudaStatus;

    // Setup the kernel dimensions
    int blockDim = sqrt(deviceProp.maxThreadsPerBlock);
    auto blockSize = dim3(blockDim, blockDim);
    //Waters are chosen for x dimension, since CUDA can handle MUCH more data along the x dimension than y.
    auto gridSize = dim3(round((blockDim - 1 + nWaters) / blockDim), round((blockDim - 1 + nDonors) / blockDim));

    // Allocate GPU buffers for vectors.
    cudaStatus = hipMalloc((void**)&dev_out, nDonors * nWaters * sizeof(char));
    if (cudaStatus != hipSuccess) {
        cerr << "hipMalloc failed!" << endl;
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_donor, nDonors * sizeof(GPUAtom));
    if (cudaStatus != hipSuccess) {
        cerr << "hipMalloc failed!" << endl;
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_linker, nLinkers * sizeof(GPUAtom));
    if (cudaStatus != hipSuccess) {
        cerr << "hipMalloc failed!" << endl;
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_water, nWaters * sizeof(GPUAtom));
    if (cudaStatus != hipSuccess) {
        cerr << "hipMalloc failed!" << endl;
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_out, out, nDonors * nWaters * sizeof(char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed!" << endl;
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_donor, inDonor, nDonors * sizeof(GPUAtom), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed!" << endl;
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_linker, inLinker, nLinkers * sizeof(GPUAtom), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed!" << endl;
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_water, inWater, nWaters * sizeof(GPUAtom), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed!" << endl;
        goto Error;
    }

    // Launch a kernel on the GPU.
    donorToWaterKernel << <gridSize, blockSize >> > (dev_out, dev_donor, dev_linker, dev_water, nDonors, nLinkers, nWaters);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        cerr << "Water to acceptor angle kernel launch failed: " << hipGetErrorString(cudaStatus) << endl;
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        cerr << "hipDeviceSynchronize returned error code " << cudaStatus << " after launching water to acceptor angle kernel!" << endl;
        cout << "Cuda failure " << __FILE__ << ":" << __LINE__ << " '" << hipGetErrorString(cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(out, dev_out, nDonors * nWaters * sizeof(char), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed!" << endl;
        goto Error;
    }

    // clear all our device arrays
Error:
    hipFree(dev_donor);
    hipFree(dev_linker);
    hipFree(dev_water);
    hipFree(dev_out);

    return cudaStatus;
}
